#include "hip/hip_runtime.h"
//===--- omptarget-nvptx.cu - NVPTX OpenMP GPU initialization ---- CUDA -*-===//
//
//                     The LLVM Compiler Infrastructure
//
// This file is dual licensed under the MIT and the University of Illinois Open
// Source Licenses. See LICENSE.txt for details.
//
//===----------------------------------------------------------------------===//
//
// This file contains the initialization code for the GPU
//
//===----------------------------------------------------------------------===//

#include "omptarget-nvptx.h"

////////////////////////////////////////////////////////////////////////////////
// global data tables
////////////////////////////////////////////////////////////////////////////////

extern __device__
    omptarget_nvptx_Queue<omptarget_nvptx_ThreadPrivateContext, OMP_STATE_COUNT>
        omptarget_nvptx_device_State[MAX_SM];

extern __device__ omptarget_nvptx_Queue<
    omptarget_nvptx_SimpleThreadPrivateContext, OMP_STATE_COUNT>
    omptarget_nvptx_device_simpleState[MAX_SM];

extern __device__ __shared__ void *omptarget_nvptx_simpleGlobalData;

////////////////////////////////////////////////////////////////////////////////
// init entry points
////////////////////////////////////////////////////////////////////////////////

INLINE unsigned nsmid() {
  unsigned n;
  asm("mov.u32 %0, %%nsmid;" : "=r"(n));
  return n;
}

INLINE unsigned smid() {
  unsigned id;
  asm("mov.u32 %0, %%smid;" : "=r"(id));
  ASSERT0(LT_FUSSY, nsmid() <= MAX_SM,
          "Expected number of SMs is less than reported.");
  return id;
}

EXTERN void __kmpc_kernel_init_params(void *Ptr) {
  PRINT(LD_IO, "call to __kmpc_kernel_init_params with version %f\n",
        OMPTARGET_NVPTX_VERSION);

  SetTeamsReductionScratchpadPtr(Ptr);
}

EXTERN void __kmpc_kernel_init(int ThreadLimit, int16_t RequiresOMPRuntime) {
  PRINT(LD_IO, "call to __kmpc_kernel_init with version %f\n",
        OMPTARGET_NVPTX_VERSION);
  ASSERT0(LT_FUSSY, RequiresOMPRuntime,
          "Generic always requires initialized runtime.");
  setExecutionParameters(Generic, RuntimeInitialized);

  int threadIdInBlock = GetThreadIdInBlock();
  ASSERT0(LT_FUSSY, threadIdInBlock == GetMasterThreadID(),
          "__kmpc_kernel_init() must be called by team master warp only!");
  PRINT0(LD_IO, "call to __kmpc_kernel_init for master\n");

  // Get a state object from the queue.
  int slot = smid() % MAX_SM;
  omptarget_nvptx_threadPrivateContext =
      omptarget_nvptx_device_State[slot].Dequeue();
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700
  omptarget_nvptx_threadPrivateContext->SetSourceQueue(slot);
#endif

  // init thread private
  int threadId = GetLogicalThreadIdInBlock();
  omptarget_nvptx_threadPrivateContext->InitThreadPrivateContext(threadId);

  // init team context
  omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
  currTeamDescr.InitTeamDescr();
  // this thread will start execution... has to update its task ICV
  // to point to the level zero task ICV. That ICV was init in
  // InitTeamDescr()
  omptarget_nvptx_threadPrivateContext->SetTopLevelTaskDescr(
      threadId, currTeamDescr.LevelZeroTaskDescr());

  // set number of threads and thread limit in team to started value
  omptarget_nvptx_TaskDescr *currTaskDescr =
      omptarget_nvptx_threadPrivateContext->GetTopLevelTaskDescr(threadId);
  currTaskDescr->NThreads() = GetNumberOfWorkersInTeam();
  currTaskDescr->ThreadLimit() = ThreadLimit;
}

EXTERN void __kmpc_kernel_deinit(int16_t IsOMPRuntimeInitialized) {
  ASSERT0(LT_FUSSY, IsOMPRuntimeInitialized,
          "Generic always requires initialized runtime.");
  // Enqueue omp state object for use by another team.
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700
  int slot = omptarget_nvptx_threadPrivateContext->GetSourceQueue();
#else
  int slot = smid() % MAX_SM;
#endif
  omptarget_nvptx_device_State[slot].Enqueue(
      omptarget_nvptx_threadPrivateContext);
  // Done with work.  Kill the workers.
  omptarget_nvptx_workFn = 0;
}

EXTERN void __kmpc_spmd_kernel_init(int ThreadLimit, int16_t RequiresOMPRuntime,
                                    int16_t RequiresDataSharing) {
  PRINT0(LD_IO, "call to __kmpc_spmd_kernel_init\n");

  if (!RequiresOMPRuntime) {
    // If OMP runtime is not required don't initialize OMP state.
    setExecutionParameters(Spmd, RuntimeUninitialized);
    if (GetThreadIdInBlock() == 0) {
      int slot = smid() % MAX_SM;
      omptarget_nvptx_simpleThreadPrivateContext =
          omptarget_nvptx_device_simpleState[slot].Dequeue();
      // Reuse the memory allocated for the full runtime as the preallocated
      // global memory buffer for the lightweight runtime.
      omptarget_nvptx_simpleGlobalData =
          omptarget_nvptx_device_State[slot].Dequeue();
    }
    __syncthreads();
    omptarget_nvptx_simpleThreadPrivateContext->Init();
    return;
  }
  setExecutionParameters(Spmd, RuntimeInitialized);

  //
  // Team Context Initialization.
  //
  // In SPMD mode there is no master thread so use any cuda thread for team
  // context initialization.
  int threadId = GetThreadIdInBlock();
  if (threadId == 0) {
    // Get a state object from the queue.
    int slot = smid() % MAX_SM;
    omptarget_nvptx_threadPrivateContext =
        omptarget_nvptx_device_State[slot].Dequeue();

    omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
    omptarget_nvptx_WorkDescr &workDescr = getMyWorkDescriptor();
    // init team context
    currTeamDescr.InitTeamDescr();
  }
  __syncthreads();

  omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
  omptarget_nvptx_WorkDescr &workDescr = getMyWorkDescriptor();

  //
  // Initialize task descr for each thread.
  //
  omptarget_nvptx_TaskDescr *newTaskDescr =
      omptarget_nvptx_threadPrivateContext->Level1TaskDescr(threadId);
  ASSERT0(LT_FUSSY, newTaskDescr, "expected a task descr");
  newTaskDescr->InitLevelOneTaskDescr(ThreadLimit,
                                      currTeamDescr.LevelZeroTaskDescr());
  newTaskDescr->ThreadLimit() = ThreadLimit;
  // install new top descriptor
  omptarget_nvptx_threadPrivateContext->SetTopLevelTaskDescr(threadId,
                                                             newTaskDescr);

  // init thread private from init value
  PRINT(LD_PAR,
        "thread will execute parallel region with id %d in a team of "
        "%d threads\n",
        newTaskDescr->ThreadId(), newTaskDescr->ThreadsInTeam());

  if (RequiresDataSharing && threadId % WARPSIZE == 0) {
    // Warp master innitializes data sharing environment.
    unsigned WID = threadId / WARPSIZE;
    __kmpc_data_sharing_slot *RootS = currTeamDescr.RootS(
        WID, WID == WARPSIZE - 1);
    DataSharingState.SlotPtr[WID] = RootS;
    DataSharingState.StackPtr[WID] = (void *)&RootS->Data[0];
  }
}

EXTERN void __kmpc_spmd_kernel_deinit() {
  // We're not going to pop the task descr stack of each thread since
  // there are no more parallel regions in SPMD mode.
  __syncthreads();
  int threadId = GetThreadIdInBlock();
  if (isRuntimeUninitialized()) {
    if (threadId == 0) {
      // Enqueue omp state object for use by another team.
      int slot = smid() % MAX_SM;
      omptarget_nvptx_device_simpleState[slot].Enqueue(
          omptarget_nvptx_simpleThreadPrivateContext);
      // Enqueue global memory back.
      omptarget_nvptx_device_State[slot].Enqueue(
          reinterpret_cast<omptarget_nvptx_ThreadPrivateContext *>(
              omptarget_nvptx_simpleGlobalData));
    }
    return;
  }
  if (threadId == 0) {
    // Enqueue omp state object for use by another team.
    int slot = smid() % MAX_SM;
    omptarget_nvptx_device_State[slot].Enqueue(
        omptarget_nvptx_threadPrivateContext);
  }
}

// Return true if the current target region is executed in SPMD mode.
EXTERN int8_t __kmpc_is_spmd_exec_mode() {
  return isSPMDMode();
}
